#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK 32

static texture<float, hipTextureType2D, hipReadModeElementType> gTex;

/**
 * Rotates an image
 */
__global__ void transform(float *dst, int w, int h, float a)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= w || y >= h)
    return;

  float u = x / (float)w;
  float v = y / (float)h;

  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(a) - v * sinf(a) + 0.5f;
  float tv = v * cosf(a) + u * sinf(a) + 0.5f;

  dst[y*w+x] = tex2D(gTex, tu, tv);
}

void process(const float *src, float *dst, int w, int h, float a)
{
  hipChannelFormatDesc chan_desc = hipCreateChannelDesc<float>();
  hipArray *cu_array;
  hipMallocArray(&cu_array, &chan_desc, w, h);
  hipMemcpyToArray(cu_array, 0, 0, src, w*h*sizeof(float), hipMemcpyHostToDevice);

  gTex.addressMode[0] = hipAddressModeBorder; 
  gTex.addressMode[1] = hipAddressModeBorder;
  gTex.filterMode = hipFilterModeLinear;
  gTex.normalized = true;

  hipBindTextureToArray(gTex, cu_array, chan_desc);
  float *output;
  hipMalloc(&output, w*h*sizeof(float));
  dim3 dimBlock(BLOCK, BLOCK);
  dim3 dimGrid((w + dimBlock.x - 1) / dimBlock.x,
               (h + dimBlock.y - 1) / dimBlock.y);

  transform<<<dimGrid, dimBlock>>>(output, w, h, a);
  hipMemcpy(dst, output, w*h*sizeof(float), hipMemcpyDeviceToHost);

  hipFreeArray(cu_array);
  hipFree(output);
}
