#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "demosaic.h"

#define BLOCK 32

namespace gpu {
static texture<float, hipTextureType2D, hipReadModeElementType> gTex;
Demosaic *Demosaic::sInstance = 0x0;

void Demosaic::Initialize() 
{
  chan_desc = hipCreateChannelDesc<float>();
}

Demosaic::~Demosaic()
{
  printf("Destroyed Demosaic\n");
}

__global__ void transform(float *dst, int w, int h, float a)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= w || y >= h)
    return;

  float u = x / (float)w;
  float v = y / (float)h;

  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(a) - v * sinf(a) + 0.5f;
  float tv = v * cosf(a) + u * sinf(a) + 0.5f;

  dst[y*w+x] = tex2D(gTex, tu, tv);
}

void Demosaic::Process(const float *src, float *dst, int w, int h, float a)
{
  hipMallocArray(&cu_array, &chan_desc, w, h);
  hipMemcpyToArray(cu_array, 0, 0, src, w*h*sizeof(float), hipMemcpyHostToDevice);

  gTex.addressMode[0] = hipAddressModeBorder; 
  gTex.addressMode[1] = hipAddressModeBorder;
  gTex.filterMode = hipFilterModeLinear;
  gTex.normalized = true;

  hipBindTextureToArray(gTex, cu_array, chan_desc);
  float *output;
  hipMalloc(&output, w*h*sizeof(float));
  dim3 dimBlock(BLOCK, BLOCK);
  dim3 dimGrid((w + dimBlock.x - 1) / dimBlock.x,
               (h + dimBlock.y - 1) / dimBlock.y);

  transform<<<dimGrid, dimBlock>>>(output, w, h, a);
  hipMemcpy(dst, output, w*h*sizeof(float), hipMemcpyDeviceToHost);

  hipFreeArray(cu_array);
  hipFree(output);
}
} // namespace gpu
