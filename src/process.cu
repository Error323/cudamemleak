#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include "init.cuh"

namespace gpu
{
  int memOp(int *data, int n)
  {
    int *d_data;

    size_t data_size = n * sizeof(int);
    cudaSafeCall(hipMalloc((void**)&d_data, data_size));

    cudaSafeCall(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));
    cudaSafeCall(hipDeviceSynchronize());

    for (int i = 0; i < 1000000; i++)
    {
      cudaSafeCall(hipMemcpy(data, d_data, data_size, hipMemcpyDeviceToHost));
      cudaSafeCall(hipDeviceSynchronize());
    }

    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipDeviceSynchronize());

    return 0;
  }
} // namespace gpu



